#include <vector>
#include <stdio.h>

#include "boost/thread/thread.hpp"

#include "gtest/gtest.h"

#include "caffe/common.hpp"
#include "caffe/syncedmem.hpp"
#include "caffe/util/device_alternate.hpp"
#include "caffe/util/math_functions.hpp"

#include "caffe/test/test_caffe_main.hpp"

namespace caffe {

class MyTest : public ::testing::Test{};

TEST_F(MyTest, TestAsyncRecycle_atGPU) {
  hipStream_t stream0;
  hipStreamCreate(&stream0);
  Caffe::set_mode(Caffe::GPU);
  SyncedMemory mem(100000000);
  SyncedMemory mem1(100000000);
  printf("%d\n", Caffe::mode() == Caffe::GPU);
  void * cpu_data = mem.mutable_cpu_data();
  caffe_memset(mem.size(), 1, cpu_data);
  cpu_data = mem1.mutable_cpu_data();
  caffe_memset(mem1.size(), 1, cpu_data);
  const void * gpu_data = mem.gpu_data();

  hipDeviceSynchronize();
  mem.mutable_gpu_data();
#pragma omp parallel sections
  {
#pragma omp section
	  {
		  mem.recycle_gpu_data(stream0);
	  }
#pragma omp section
	  {
		  mem1.gpu_data();
	  }
  }
  hipDeviceSynchronize();
}

TEST_F(MyTest, TestAsyncRecycle_Synced) {
  hipStream_t stream0;
  hipStreamCreate(&stream0);
  Caffe::set_mode(Caffe::GPU);
  SyncedMemory mem(100000000);
  SyncedMemory mem1(100000000);
  printf("%d\n", Caffe::mode() == Caffe::GPU);
  void * cpu_data = mem.mutable_cpu_data();
  caffe_memset(mem.size(), 1, cpu_data);
  cpu_data = mem1.mutable_cpu_data();
  caffe_memset(mem1.size(), 1, cpu_data);
  const void * gpu_data = mem.gpu_data();

  hipDeviceSynchronize();
#pragma omp parallel sections
  {
#pragma omp section
	  {
		  mem.recycle_gpu_data(stream0);
	  }
#pragma omp section
	  {
		  mem1.gpu_data();
	  }
  }
  hipDeviceSynchronize();
}

}
